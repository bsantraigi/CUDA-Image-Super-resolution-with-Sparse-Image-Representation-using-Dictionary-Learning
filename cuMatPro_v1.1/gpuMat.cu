#pragma once

#include "hip/hip_runtime.h"


#include <iostream>

#define pn(x) printf("%5.2f ", (double)x)

using namespace std;

template <typename T>
class gpuMat
{
public:
	T* h_elems = nullptr;
	T* d_elems = nullptr;
	int rows, cols;

	gpuMat();
	gpuMat(int rows, int cols);
	~gpuMat();
	T& operator()(int row, int col = 0);
	void print(bool start = true);
	void copy2Device();
	void copy2Host();
};

template <typename T>
gpuMat<T>::gpuMat()
{	
}

template <typename T>
gpuMat<T>::gpuMat(int rows, int cols)
{
	this->rows = rows;
	this->cols = cols;
	h_elems = new T[rows*cols];
	hipMalloc(&d_elems, rows*cols*sizeof(double));
}

template <typename T>
gpuMat<T>::~gpuMat()
{
	cout << "Destroying gpuMat[auto]" << endl;
	delete[] h_elems;
	hipFree(d_elems);

}

template <typename T>
T& gpuMat<T>::operator()(int row, int col)
{
	return h_elems[col*rows + row];
}

template <typename T>
void gpuMat<T>::copy2Device()
{
	hipMemcpy(d_elems, h_elems, rows*cols*sizeof(T), hipMemcpyHostToDevice);
}

template <typename T>
void gpuMat<T>::copy2Host()
{
	hipMemcpy(h_elems, d_elems, rows*cols*sizeof(T), hipMemcpyDeviceToHost);
}

template <typename T>
void gpuMat<T>::print(bool start)
{
	cout << endl;
	cout << start << " <- start" << endl;
	if (start){
		for (int i = 0; i < min(10, rows); i++)
		{
			for (int j = 0; j < min(10, cols); j++)
			{
				pn((*this)(i, j));
			}
			cout << endl;
		}
	}
	else{
		for (int i = max(0, rows - 10); i < rows; i++)
		{
			for (int j = max(10, cols - 10); j < cols; j++)
			{
				pn((*this)(i, j));
			}
			cout << endl;
		}
	}
}