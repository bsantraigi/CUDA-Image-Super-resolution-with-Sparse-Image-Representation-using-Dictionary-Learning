#pragma once

#include "hip/hip_runtime.h"


#include <iostream>

#define pn(x) printf("%6.3f ", (double)x)

using namespace std;

template <typename T>
class gpuMat
{
	bool blank = true;
public:
	T* h_elems = nullptr;
	T* d_elems = nullptr;
	int rows, cols;
	int2 *d_size;

	gpuMat();
	gpuMat(int rows, int cols);
	void create(int rows, int cols);
	~gpuMat();
	T& operator()(int row, int col = 0);
	void print(bool start = true);
	void copy2Device();
	void copy2Host();
	void ToFile(string fileName);
private:
	void SetSize();
};

template <typename T>
gpuMat<T>::gpuMat()
{
	blank = true;
}

template <typename T>
void gpuMat<T>::SetSize()
{
	// Initialize d_size
	int2 h_size;
	h_size.x = cols;
	h_size.y = rows;
	hipMalloc(&d_size, sizeof(int2));
	hipMemcpy(d_size, &h_size, sizeof(int2), hipMemcpyHostToDevice);
}
template <typename T>
gpuMat<T>::gpuMat(int rows, int cols)
{
	create(rows, cols);
}


template <typename T>
void gpuMat<T>::create(int rows, int cols)
{
	if (!blank){
		delete[] h_elems;
		hipFree(d_elems);
	}
	blank = false;
	this->rows = rows;
	this->cols = cols;
	h_elems = new T[rows*cols];
	hipError_t err = hipMalloc(&d_elems, rows*cols*sizeof(double));
	if (err != hipSuccess){
		cout << "[gpuMat::ctor]Memory allocation on GPU failed." << endl;
	}

	SetSize();
}

template <typename T>
gpuMat<T>::~gpuMat()
{
	if (!blank){
		cout << "[gpuMat::dtor]Destroying gpuMat[auto]" << endl;
		delete[] h_elems;
		hipFree(d_elems);
	}
	else{
		cout << "[gpuMat::dtor] object was blank" << endl;
	}
	hipFree(d_size);
}

template <typename T>
T& gpuMat<T>::operator()(int row, int col)
{
	return h_elems[col*rows + row];
}

template <typename T>
void gpuMat<T>::copy2Device()
{
	hipMemcpy(d_elems, h_elems, rows*cols*sizeof(T), hipMemcpyHostToDevice);
}

template <typename T>
void gpuMat<T>::copy2Host()
{
	hipMemcpy(h_elems, d_elems, rows*cols*sizeof(T), hipMemcpyDeviceToHost);
}

template <typename T>
void gpuMat<T>::print(bool start)
{
	// set start = false if you want to print the bottom-right corner
	// of a large matrix
	cout << endl;
	if (start){
		for (int i = 0; i < min(10, rows); i++)
		{
			for (int j = 0; j < min(10, cols); j++)
			{
				pn((*this)(i, j));
			}
			cout << endl;
		}
	}
	else{
		for (int i = max(0, rows - 10); i < rows; i++)
		{
			for (int j = max(10, cols - 10); j < cols; j++)
			{
				pn((*this)(i, j));
			}
			cout << endl;
		}
	}
}

template <typename T>
void gpuMat<T>::ToFile(string filename)
{
	this->copy2Host();
	FILE *fh;
	fh = fopen(filename.c_str(), "w");
	if (fh != NULL){
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
			{
				fprintf(fh, "%6.2f", (*this)(i, j));
				//os << (*this)(i, j) << ",";
				//os << j*rows + i << ",";
				//os << "[" << j*rows + i  << "]" << h_elems[j*rows + i] << ",";
			}
			//os << endl;
			fprintf(fh, "\n");
		}
		fclose(fh);
	}
	else{
		cout << filename << " failed to open..." << endl;
	}
}