#pragma once

#include "hip/hip_runtime.h"


#include <iostream>

#define pn(x) printf("%5.2f ", (double)x)

using namespace std;

template <typename T>
class gpuMat
{
	bool blank = true;
public:
	T* h_elems = nullptr;
	T* d_elems = nullptr;
	int rows, cols;

	gpuMat();
	gpuMat(int rows, int cols);
	void create(int rows, int cols);
	~gpuMat();
	T& operator()(int row, int col = 0);
	void print(bool start = true);
	void copy2Device();
	void copy2Host();
};

template <typename T>
gpuMat<T>::gpuMat()
{
	blank = true;
}

template <typename T>
gpuMat<T>::gpuMat(int rows, int cols)
{
	if (!blank){
		delete[] h_elems;
		hipFree(d_elems);
	}
	blank = false;	
	this->rows = rows;
	this->cols = cols;
	h_elems = new T[rows*cols];
	hipError_t err = hipMalloc(&d_elems, rows*cols*sizeof(double));
	if (err != hipSuccess){
		cout << "[gpuMat::ctor]Memory allocation on GPU failed." << endl;
	}
}

template <typename T>
void gpuMat<T>::create(int rows, int cols)
{
	if (!blank){
		delete[] h_elems;
		hipFree(d_elems);
	}
	blank = false;
	this->rows = rows;
	this->cols = cols;
	h_elems = new T[rows*cols];
	hipError_t err = hipMalloc(&d_elems, rows*cols*sizeof(double));
	if (err != hipSuccess){
		cout << "[gpuMat::ctor]Memory allocation on GPU failed." << endl;
	}
}

template <typename T>
gpuMat<T>::~gpuMat()
{
	if (!blank){
		cout << "[gpuMat::dtor]Destroying gpuMat[auto]" << endl;
		delete[] h_elems;
		hipFree(d_elems);
	}
	else{
		cout << "[gpuMat::dtor] object was blank" << endl;
	}

}

template <typename T>
T& gpuMat<T>::operator()(int row, int col)
{
	return h_elems[col*rows + row];
}

template <typename T>
void gpuMat<T>::copy2Device()
{
	hipMemcpy(d_elems, h_elems, rows*cols*sizeof(T), hipMemcpyHostToDevice);
}

template <typename T>
void gpuMat<T>::copy2Host()
{
	hipMemcpy(h_elems, d_elems, rows*cols*sizeof(T), hipMemcpyDeviceToHost);
}

template <typename T>
void gpuMat<T>::print(bool start)
{
	cout << endl;
	cout << start << " <- start" << endl;
	if (start){
		for (int i = 0; i < min(10, rows); i++)
		{
			for (int j = 0; j < min(10, cols); j++)
			{
				pn((*this)(i, j));
			}
			cout << endl;
		}
	}
	else{
		for (int i = max(0, rows - 10); i < rows; i++)
		{
			for (int j = max(10, cols - 10); j < cols; j++)
			{
				pn((*this)(i, j));
			}
			cout << endl;
		}
	}
}