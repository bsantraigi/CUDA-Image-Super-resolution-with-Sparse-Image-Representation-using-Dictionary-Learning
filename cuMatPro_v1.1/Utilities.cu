#include "Utilities.cu.h"

map<string, int> Utilities::groupCounts;
map<string, vector<string> > Utilities::groupedImages;

Utilities::Utilities()
{
}

void Utilities::DisplayMat(Mat &image, string s)
{
	cout << s << " of size " << image.rows << ", " << image.cols << endl;
	if (s.size() == 0) {
		namedWindow("Display window", WINDOW_AUTOSIZE); // Create a window for display.
		imshow("Display window", image); // Show our image inside it.
	}
	else {
		namedWindow(s, WINDOW_AUTOSIZE); // Create a window for display.
		imshow(s, image); // Show our image inside it.
	}
}

vector<string> Utilities::GetAllFiles(string imageCategory)
{	
	if (groupCounts.size() == 0) {
		cout << "Generating file list..." << endl;
		ifstream inf("rfiles/pointerList.list1");
		char a[80];
		while (!inf.eof()) {
			inf >> a;
			string cat(a);
			if (cat.size() == 0) {
				break;
			}
			inf >> a;
			groupCounts[cat] = atoi(a);
			
			ifstream catFileStream("rfiles/" + cat + ".list2");
			groupedImages[cat] = vector<string>();
			char b[1024];
			while (!catFileStream.eof()) {
				catFileStream >> b;
				if (strlen(b) == 0) {
					break;
				}
				groupedImages[cat].push_back(string(b));
			}
			catFileStream.close();
		}
		inf.close();
	}
	
	return groupedImages[imageCategory];	
}

void Utilities::prettyStart(string s)
{
	cout << endl << "***************************************" << endl;
	cout << "\t" << s << endl;
	cout << "***************************************" << endl << endl;
}
void Utilities::prettyEnd(string s)
{
	cout << endl << "=======================================" << endl;
	cout << "\t" << s << endl;
	cout << "=======================================" << endl << endl;
}

Utilities::~Utilities()
{
}
