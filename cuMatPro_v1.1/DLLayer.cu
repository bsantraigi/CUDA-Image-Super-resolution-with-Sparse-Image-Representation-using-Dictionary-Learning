#include "hip/hip_runtime.h"
#pragma once

// Macro for timing kernel runs
#define START_METER {\
	hipEvent_t start, stop;\
	float elapsedTime;\
	hipEventCreate(&start);\
	hipEventRecord(start, 0);
#define STOP_METER hipEventCreate(&stop);\
	hipEventRecord(stop, 0);\
	hipEventSynchronize(stop);\
	hipEventElapsedTime(&elapsedTime, start, stop);\
	printf("Elapsed time : %f ms\n", elapsedTime);\
	}

//Do kernel activity here
// Standard/CUDA Includes
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <hipsolver.h>

// User defined Libraries
#include "Utilities.cu.h"
#include "gpuMat.cu"
#include "gpuOpsAPI.cu"

using namespace std;

/*
DATA STRUCTURES
*/
typedef struct {
	double gam_d, gam_s, gam_n, gam_bias;
} _modelParams;

typedef struct{
	int K;
	double a_d, a_s, a_bias, a_n;
	double b_d, b_s, b_bias, b_n;
	double a_pi, b_pi;
} _dlConfig;

/*
* DATA STRUCT: _cusolverStruct
*	This data structure contains matrices and helper cuda variables
*	required for sampling of D.col or S.col from a Multivariate Random distribution
*/
typedef struct{
	hipsolverHandle_t handle; //Host
	int Lwork; // Host
	double* workspace; //Device
	int *devInfo; // Device
	int2* d_size;

	// Host-Device paired
	gpuMat<double> mu;
	gpuMat<double> covar;

	gpuMat<double> eigenVals; 
	gpuMat<double> diagonally;
	gpuMat<double> L;
} _cusolverStruct;

// GPU Kernels

/*
DEFINITIONS
*/
__global__ void setup_kernel(hiprandState_t* d_localstates, unsigned int seed)
{
	/*QUALIFIERS void hiprand_init(unsigned long long seed,
	unsigned long long subsequence,
	unsigned long long offset,
	hiprandStateXORWOW_t *state)*/
	int id = threadIdx.x;
	hiprand_init(seed, id, 0, &d_localstates[id]);
}

/*
Gamma Random Variable generator
Marsaglia and Tsang�s Method
*/
__device__ void gamrnd_d(double* x, double2* params, hiprandState_t* d_localstates)
{
	double alpha = params->x;
	double beta = params->y;

	if (alpha >= 1){
		hiprandState_t localState = *d_localstates; // Be careful the change in localState variable needs to be reflected back to d_localStates
		double d = alpha - 1 / 3.0, c = 1 / sqrt(9 * d);
		do{
			double z = hiprand_normal(&localState);
			double u = hiprand_uniform(&localState);
			double v = pow((double) 1.0f + c*z, (double) 3.0f);
			if (z > -1 / c && log(u) < (z*z / 2 + d - d*v + d*log(v))){
				*x = d*v / beta;
				*d_localstates = localState;
				//printf("GRND: a = %f, b = %f, x = %f\n", alpha, beta, *x);
				return;
			}
		} while (true);
	}
	else{
		double r;
		params->x += 1;
		gamrnd_d(&r, params, d_localstates);

		hiprandState_t localState = *d_localstates;
		double u = hiprand_uniform(&localState);
		*x = r*pow((double)u, (double)1 / alpha);
		params->x -= 1;
		return;
	}
}

/*
Algorithm as mentioned in Wikipedia:
x ~ Gamma(a, 1)
y ~ Gamma(b, 1)
then,
z = x/(x+y) ~ Beta(a, b)
*/
__device__ void betarnd_d(double* x, double2* params, hiprandState_t* d_localstates)
{
	double2 params1{ params->x, 1 };
	double x1;
	gamrnd_d(&x1, &params1, d_localstates);

	double2 params2{ params->y, 1 };
	double x2;
	gamrnd_d(&x2, &params2, d_localstates);

	*x = x1 / (x1 + x2);
}

__global__ void InitGammaParams_kernel(_modelParams* modelParams, _dlConfig* dlConfig, hiprandState_t* d_localstates)
{
	double2 hyperParams_d{ dlConfig->a_d, dlConfig->b_d };
	double2 hyperParams_s{ dlConfig->a_s, dlConfig->b_s };
	double2 hyperParams_n{ dlConfig->a_n, dlConfig->b_n };
	double2 hyperParams_bias{ dlConfig->a_bias, dlConfig->b_bias };
	double gam_d, gam_s, gam_n, gam_bias;
	// Sample from gamrnd_d here
	gamrnd_d(&gam_d, &hyperParams_d, d_localstates);
	gamrnd_d(&gam_s, &hyperParams_s, d_localstates);
	gamrnd_d(&gam_n, &hyperParams_n, d_localstates);
	gamrnd_d(&gam_bias, &hyperParams_bias, d_localstates);

	// Copy the sampled values to modelParams
	modelParams->gam_d = gam_d;
	modelParams->gam_s = gam_s;
	modelParams->gam_n = gam_n;
	modelParams->gam_bias = gam_bias;
}

__global__ void copyDiag_sqrt(double* srcVec, double* destMat, int m)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	destMat[i*m + i] = sqrt(srcVec[i]);
}

/*
Just fill in the D[:, col] with samples from a standard normal Dist.
Just launch with 1D launch configuration - split into blocks s.t.
each block has close to permissible amount of concurrent threads
*/
__global__ void DPointSample_kernel(double* D, int2* _size, int col, hiprandState_t* d_localstates)
{
	int Rows = _size->y;
	int Cols = _size->x;

	int row = blockIdx.x*blockDim.x + threadIdx.x;
	D[col*Rows + row] = hiprand_normal(&d_localstates[row]);
}

/*
Using LLT decomposition transform the already sampled
column D[:, col]

Launch Rows number of threads
*/
__global__ void DSetVar_kernel(double* D, double* L, int2* _size, int col)
{
	// Dimension of D
	int Rows = _size->y;
	int DCols = _size->x;

	// covar = L'*L
	// Multiply L' * D[:, col]
	// While multiplying use only the lower part of the covar matrix
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	double factor = D[col*DCols + k];
	for (int i = 0; i < Rows; i++)
	{
		L[k*Rows + i] *= factor;
	}

	
	double cValue = 0;
	for (int j = 0; j < Rows; j++)
	{
		cValue += L[j*Rows + k];
	}
	D[col*DCols + k] = cValue;
}

__global__ void DSetMean_kernel(double* D, double* muD, int2* _size, int col)
{
	int Rows = _size->y;
	int Cols = _size->x;

	int row = blockIdx.x*blockDim.x + threadIdx.x;
	D[col*Rows + row] = D[col*Rows + row] + muD[row];
}

/*
Calculate value of N from imsize, patchsize & imcount
*/
int calcN(int imsize, int patchsize, int imcount)
{
	return (imsize - patchsize + 1)*(imsize - patchsize + 1)*imcount;
}

void InitSolverKit_evd(_cusolverStruct &solverKit, gpuMat<double> &D)
{
	int Rows = D.rows;
	solverKit.eigenVals.create(D.rows, 1);
	solverKit.L.create(D.rows, D.rows);

	// Initialize d_size
	int2 h_size{ D.rows, D.cols };
	hipMalloc(&solverKit.d_size, sizeof(int2));
	hipMemcpy(solverKit.d_size, &h_size, sizeof(int2), hipMemcpyHostToDevice);

	// Initialize covariance matrix to I
	solverKit.covar.create(D.rows, D.rows);
	gpuMat<double> &covar = solverKit.covar;
	for (int i = 0; i < D.rows; i++)
	{
		for (int j = 0; j < D.rows; j++)
		{
			covar(i, j) = (i == j) ? 1 : 0;
		}
	}
	covar.copy2Device();

	// Initialize diagonally matrix
	solverKit.diagonally.create(D.rows, D.rows);
	gpuMat<double> &diagonally = solverKit.diagonally;
	for (int i = 0; i < D.rows; i++)
	{
		for (int j = 0; j < D.rows; j++)
		{
			diagonally(i, j) = 0;
		}
	}
	diagonally.copy2Device();

	// Initialize mu vector
	solverKit.mu.create(Rows, 1);
	gpuMat<double> &mu = solverKit.mu;
	for (int i = 0; i < D.rows; i++)
	{
		mu(i, 0) = 0;
	}
	mu.copy2Device();

	hipblasFillMode_t MODE = HIPBLAS_FILL_MODE_UPPER;
	hipsolverDnCreate(&solverKit.handle);
	hipMalloc(&solverKit.devInfo, sizeof(int));

	hipsolverDnDsyevd_bufferSize(solverKit.handle, HIPSOLVER_EIG_MODE_VECTOR,
		MODE, Rows, solverKit.covar.d_elems, Rows, solverKit.eigenVals.d_elems, &solverKit.Lwork);

	hipMalloc(&solverKit.workspace, solverKit.Lwork*sizeof(double));
}

/*
M = LL^T decomposition for covariance matrix
*/
void LLT_d(_cusolverStruct &solverKit)
{
	gpuMat<double> &covar = solverKit.covar;
	hipblasFillMode_t MODE = HIPBLAS_FILL_MODE_LOWER;
	hipsolverStatus_t status;
	gpuMat<double> &W = solverKit.eigenVals;
	int Rows = covar.rows;
	// cusolverDn - for dense mat, D - Double, potrf - Cholesky solver
	// sytrf - LDLT decomposition
	// syevd - EigenValue decomp
	// Taking Long time ~ 20 ms
	status = hipsolverDnDsyevd(solverKit.handle, HIPSOLVER_EIG_MODE_VECTOR, MODE,
		covar.rows, covar.d_elems, covar.rows, W.d_elems, solverKit.workspace, solverKit.Lwork, solverKit.devInfo);

	const int L = std::min(Rows, 32);
	dim3 threadsPerBlock(L);
	dim3 numBlocks((unsigned int)ceil((double)Rows / L));

	copyDiag_sqrt<<<numBlocks, threadsPerBlock>>>(W.d_elems, solverKit.diagonally.d_elems, Rows);
	MatMul<double, double, double>(covar.d_elems, solverKit.diagonally.d_elems, solverKit.L.d_elems, Rows, Rows, Rows);
	if (status != hipSuccess){
		cout << "LLT Decomp. Failed Badly !!!" << endl;
	}
}

/*
CLASS DLLayer
*/
class DLLayer
{
public:
	// Model params
	_modelParams *h_params, *d_params;
	// Keep the size of d_localstates M*M or K
	hiprandState_t *d_localstates; // Device Allocated
	int statesCount;

	// DLConfig: Layer cofigurations
	_dlConfig *h_dlConfig;
	_dlConfig *d_dlConfig;

	// Actual YDSB matrices
	int N, M, K;
	gpuMat<double> D;
	gpuMat<double> S;
	gpuMat<bool> B;
	gpuMat<double> PI;
	gpuMat<double> post_PI;

	// Solverkits for cholesky decomposition while sampling D[:, i] and S[:, j]
	_cusolverStruct solverKitD;
	_cusolverStruct solverKitS;

	DLLayer(int propImSize = 256, int propPatchSize = 8, int propImCount = 5);
	~DLLayer();

	void Init();
	void reflect();
	void mvnrnd_d(gpuMat<double> &holder, _cusolverStruct &solverKit, int col);
};

/*
DLLayer Class method definitions
Must use copy constructor for objects (no need for pointers as temporary
objects aren't the issue there)
*/
DLLayer::DLLayer(int propImSize, int propPatchSize, int propImCount) : 
M(propPatchSize*propPatchSize), N(calcN(propImSize, propPatchSize, propImCount)), K(100),
D(gpuMat<double>(M, K)), S(gpuMat<double>(K, N)), B(gpuMat<bool>(K, N)), PI(gpuMat<double>(K, 1)), post_PI(gpuMat<double>(K, N))
{
	Utilities::prettyStart("Constructing LAYER");
	// DLLayer Matrices YDSB	
	cout << "M: " << M << ", N: " << N << ", K: " << K << endl;
	statesCount = max(K, M);

	// Model Params ctor
	h_params = new _modelParams();
	hipMalloc(&d_params, sizeof(_modelParams));
	
	hipMalloc(&d_localstates, statesCount*sizeof(hiprandState_t));
	// Need only a single state variable
	setup_kernel << <(uint)ceil(statesCount/32), 32 >> >(d_localstates, (unsigned int)time(NULL));

	// DLConfig.cu
	h_dlConfig = new _dlConfig();
	h_dlConfig->K = 80;
	h_dlConfig->a_d = h_dlConfig->a_s = h_dlConfig->a_bias = 1;
	h_dlConfig->b_d = h_dlConfig->b_s = h_dlConfig->b_bias = 1;
	h_dlConfig->a_n = h_dlConfig->b_n = 1e-1;

	h_dlConfig->a_pi = 1;
	h_dlConfig->b_pi = 1200;

	hipMalloc(&d_dlConfig, sizeof(_dlConfig));
	hipMemcpy(d_dlConfig, h_dlConfig, sizeof(_dlConfig), hipMemcpyHostToDevice);
	
	Utilities::prettyEnd("LAYER Constructed");	

	Utilities::prettyStart("Layer Initialization STARTING");
	this->Init();
	Utilities::prettyStart("Layer Initialization Complete");
}


DLLayer::~DLLayer()
{
	cout << "DLLayer destructor!!!" << endl;

	//ModelParams.cu
	cout << "Destroying Model Params" << endl;
	delete[] h_params;
	hipFree(d_params);
	hipFree(d_localstates);

	// DLConfig.cu
	cout << "Destroying dlConfig object" << endl;
	free(h_dlConfig);
	hipFree(d_dlConfig);

	
}

void DLLayer::Init()
{
	// Initialize Model hyperparameters - Generating initial samples of gammas
	InitGammaParams_kernel <<<1, 1 >>>(d_params, d_dlConfig, d_localstates);
	//this->reflect(); // Required if printing the values

	// Initialize LLT transformation sovler Kit
	InitSolverKit_evd(solverKitD, D);
	InitSolverKit_evd(solverKitS, S);

	// Sample columns of D
	cout << "** SAMPLING COLUMNS of D **" << endl;
	
	for (int k = 0; k < K; k++)
	{
		mvnrnd_d(D, solverKitD, k);
		break;
	}

		
	
}

void DLLayer::reflect()
{
	// reflect current state of hyperparams
	hipMemcpy(h_params, d_params, sizeof(_modelParams), hipMemcpyDeviceToHost);
}

void DLLayer::mvnrnd_d(gpuMat<double> &holder, _cusolverStruct &solverKit, int col)
{
	gpuMat<double> &covar = solverKit.covar;
	int Rows = holder.rows;
	int Cols = holder.cols;

	const int L = std::min(Rows, 32);
	dim3 threadsPerBlock(L);
	dim3 numBlocks((unsigned int)ceil((double)Rows / L));

	
	// Launch DPointSample_kernel
	DPointSample_kernel <<<numBlocks, threadsPerBlock>>> (holder.d_elems, solverKit.d_size, col, d_localstates);

	// LLT Decomp of covar and find Ld	
	// Takes most time - 21 ms for a single call
	LLT_d(solverKitD);

	// Apply covar transformation - Multiply with Ld 
	// Taking SECOND MOST time
	DSetVar_kernel<<<numBlocks, threadsPerBlock>>>(D.d_elems, solverKit.L.d_elems, solverKit.d_size, col);

	// Add mu - Launch DSetMean_kernel
	DSetMean_kernel<<<numBlocks, threadsPerBlock>>>(D.d_elems, solverKit.mu.d_elems, solverKit.d_size, col);
}

