#pragma once

#include "ModelParams_kernels.cu"

using namespace std;

class ModelParams{
public:
	_modelParams *h;
	_modelParams *d;
	hiprandState_t *localState;

	ModelParams();
	~ModelParams();
	void Init(DLConfig &dlConfig);
	void reflect();
};

ModelParams::ModelParams()
{
	h = new _modelParams();
	hipMalloc(&d, sizeof(_modelParams));
	hipMalloc(&localState, sizeof(hiprandState_t));
	// Need only a single state variable
	setup_kernel << <1, 1 >> >(localState, time(NULL));
}

ModelParams::~ModelParams()
{
	cout << "Destroying Model Params" << endl;
	delete[] h;
	hipFree(d);
	hipFree(localState);
}

void ModelParams::Init(DLConfig &dlConfig)
{
	initGibbsParams(this->d, dlConfig.d, localState);
	this->reflect();
	cout << "Initial Sample gam_d: " << h->gam_d << endl;
}

void ModelParams::reflect()
{
	hipMemcpy(h, d, sizeof(_modelParams), hipMemcpyDeviceToHost);
}