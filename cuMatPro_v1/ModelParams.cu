#include "ModelParams.cu.h"

ModelParams::ModelParams()
{
	h = new _modelParams();
	hipMalloc(&d, sizeof(_modelParams));
	hipMalloc(&localState, sizeof(hiprandState_t));
	// Need only a single state variable
	setup_kernel << <1, 1 >> >(localState, time(NULL));
}

ModelParams::~ModelParams()
{
	cout << "Destroying Model Params" << endl;
	delete[] h;
	hipFree(d);
	hipFree(localState);
}

void ModelParams::Init(DLConfig &dlConfig)
{
	initGibbsParams_kernel << <1, 2 >> >(this->d, dlConfig.d, localState);
	this->reflect();
	cout << "Initial Sample gam_d: " << h->gam_d << endl;
}

void ModelParams::reflect()
{
	hipMemcpy(h, d, sizeof(_modelParams), hipMemcpyDeviceToHost);
}