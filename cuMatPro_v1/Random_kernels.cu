#include "hip/hip_runtime.h"
#include "Random_kernels.cu.h"

/*
DEFINITIONS
*/
__global__ void setup_kernel(hiprandState_t* d_localstates, unsigned int seed)
{
	/*QUALIFIERS void hiprand_init(unsigned long long seed,
	unsigned long long subsequence,
	unsigned long long offset,
	hiprandStateXORWOW_t *state)*/
	int id = threadIdx.x;
	hiprand_init(seed, id, 0, &d_localstates[id]);
}

/*
Gamma Random Variable generator
Marsaglia and Tsang�s Method
*/
__device__ void gamrnd_d(double* x, double2* params, hiprandState_t* d_localstates)
{
	double alpha = params->x;
	double beta = params->y;

	if (alpha >= 1){
		hiprandState_t localState = *d_localstates; // Be careful the change in localState variable needs to be reflected back to d_localStates
		double d = alpha - 1 / 3.0, c = 1 / sqrt(9 * d);
		do{
			double z = hiprand_normal(&localState);
			double u = hiprand_uniform(&localState);
			double v = pow((double) 1.0f + c*z, (double) 3.0f);
			double extra = 0;
			if (z > -1 / c && log(u) < (z*z / 2 + d - d*v + d*log(v))){
				*x = d*v / beta;
				*d_localstates = localState;
				return;
			}
		} while (true);
	}
	else{
		double r;
		params->x += 1;
		gamrnd_d(&r, params, d_localstates);

		hiprandState_t localState = *d_localstates;
		double u = hiprand_uniform(&localState);
		*x = r*pow((double)u, (double)1 / alpha);
		params->x -= 1;
		return;
	}
}

/*
Algorithm as mentioned in Wikipedia:
x ~ Gamma(a, 1)
y ~ Gamma(b, 1)
then,
z = x/(x+y) ~ Beta(a, b)
*/
__device__ void betarnd_d(double* x, double2* params, hiprandState_t* d_localstates)
{
	double alpha = params->x;
	double beta = params->y;

	double2 params1{ params->x, 1 };
	double x1;
	gamrnd_d(&x1, &params1, d_localstates);

	double2 params2{ params->y, 1 };
	double x2;
	gamrnd_d(&x2, &params2, d_localstates);

	*x = x1 / (x1 + x2);
}
