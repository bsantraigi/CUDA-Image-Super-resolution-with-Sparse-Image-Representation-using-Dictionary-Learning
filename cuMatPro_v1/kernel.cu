#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include "gpuMat.h"
#include "gpuOpsAPI.h"
#include "Timer.h"
#include "DLLayer_GPU.h"
#include "ImLoader.h"
#include "DLConfig.h"
#include "ModelParams.h"

#define min(x, y) ((x)<(y)?(x):(y))

using namespace std;

Timer timer1;

int test()
{
	int S = 20;
	gpuMat<float> Y(S, S);
	gpuMat<bool> B(S, S);
	gpuMat<double> C(S, S);
	cout << Y.cols << "by" << Y.rows << endl;

	for (int i = 0; i < S; i++)
	{
		for (int j = 0; j < S; j++)
		{
			Y(i, j) = i*Y.cols + j;
			B(i, j) = (i>=j);
		}
	}
	Y.copy2Device();
	B.copy2Device();

	Y.print();
	B.print();

	// CUBLAS TEST
	/*float al = 1;
	float bet = 0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, S, &al, Y.d_elems, S, B.d_elems, S, &bet, C.d_elems, S);*/

	MatMul<float, bool, double>(Y.d_elems, B.d_elems, C.d_elems, S, S, S);

	C.copy2Host();
	C.print();


	// Test functions for rectangular matrices
	int m = 682768, n = 256, k = 128;
	gpuMat<float> mat1(m, k);
	gpuMat<float> vec1(k, n);
	gpuMat<float> result(m, n);

	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < k; j++)
		{
			//mat1(i, j) = ((i + 1)%(j + 1));
			mat1(i, j) = (float)rand() / RAND_MAX - 0.5;
		}
	}

	for (int i = 0; i < k; i++)
	{
		for (int j = 0; j < n; j++)
		{
			vec1(i, j) = (float)rand() / RAND_MAX - 0.5;
		}
	}
	
	mat1.print();
	vec1.print();

	mat1.copy2Device();
	vec1.copy2Device();

	cout << "Using my API." << endl;
	{
		hipEvent_t start, stop;
		float elapsedTime;

		hipEventCreate(&start);
		hipEventRecord(start, 0);

		//Do kernel activity here
		MatMul<float, float, float>(mat1.d_elems, vec1.d_elems, result.d_elems, m, n, k);

		hipEventCreate(&stop);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsedTime, start, stop);
		printf("Elapsed time : %f ms\n", elapsedTime);
	}

	result.copy2Host();
	
	result.print();

	cout << "Using CUBLAS" << endl;
	float al = 1;
	float bet = 0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	{
		hipEvent_t start, stop;
		float elapsedTime;

		hipEventCreate(&start);
		hipEventRecord(start, 0);

		//Do kernel activity here
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &al, mat1.d_elems, m, vec1.d_elems, k, &bet, result.d_elems, m);

		hipEventCreate(&stop);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsedTime, start, stop);
		printf("Elapsed time : %f ms\n", elapsedTime);
	}
	result.copy2Host();

	result.print();

	/*cout << "Calculating in host CPU | Single thread" << endl;
	{
		timer1.start();
		for (int i = 0; i < m; i++)
		{
			for (int j = 0; j < n; j++)
			{
				double cvalue = 0;
				for (int l = 0; l < k; l++)
				{
					cvalue += mat1(i, l)*vec1(l, j);
				}
				result(i, j) = cvalue;
			}
		}
		timer1.stop();
	}

	result.print();*/

	return 0;
}

int calcN(int imsize, int patchsize, int imcount)
{
	return (imsize - patchsize + 1)*(imsize - patchsize + 1)*imcount;
}
void DLCode();
void testRand();
int main(){
	//test();
	//DLCode();
	testRand();
}

__global__ void setup_kernel(hiprandState_t* d_localstates)
{
	int id = threadIdx.x;
	hiprand_init(1234, id, 0, &d_localstates[id]);
}

__global__ void generate_kernel(double* d_samples, hiprandState_t* d_localstates)
{
	int length = 16;
	int sid = threadIdx.x * length;
	hiprandState_t localState = d_localstates[threadIdx.x];
	for (int i = 0; i < length; i++)
	{
		d_samples[sid + i] = hiprand_uniform(&localState);
	}
}

/*
Gamma Random Variable generator
Marsaglia and Tsang�s Method
*/
__device__ void gamrnd_d(double* x, double2* params, hiprandState_t* d_localstates)
{
	double alpha = params->x;
	double beta = params->y;	

	if (alpha >= 1){
		hiprandState_t localState = *d_localstates; // Be careful the change in localState variable needs to be reflected back to d_localStates
		double d = alpha - 1 / 3.0, c = 1 / sqrt(9 * d);
		do{
			double z = hiprand_normal(&localState);
			double u = hiprand_uniform(&localState);
			double v = pow((double) 1.0f + c*z, (double) 3.0f);
			double extra = 0;
			if (z > -1 / c && log(u) < (z*z / 2 + d - d*v + d*log(v))){
				*x = d*v/beta;
				*d_localstates = localState;
				return;
			}
		} while (true);
	}
	else{
		double r;
		params->x += 1;
		gamrnd_d(&r, params, d_localstates);

		hiprandState_t localState = *d_localstates;
		double u = hiprand_uniform(&localState);
		*x = r*pow((double)u, (double) 1 / alpha);
		params->x -= 1;
		return;
	}
}

/*
Algorithm as mentioned in Wikipedia:
x ~ Gamma(a, 1)
y ~ Gamma(b, 1)
then,
z = x/(x+y) ~ Beta(a, b)
*/
__device__ void betarnd_d(double* x, double2* params, hiprandState_t* d_localstates)
{
	double alpha = params->x;
	double beta = params->y;

	double2 params1{ params->x, 1 };
	double x1;
	gamrnd_d(&x1, &params1, d_localstates);

	double2 params2{ params->y, 1 };
	double x2;
	gamrnd_d(&x2, &params2, d_localstates);

	*x = x1 / (x1 + x2);
}

__global__ void gammaTest_kernel(hiprandState_t* d_localstates, double2* params, double* d_samples, int length)
{
	int sid = threadIdx.x * length;
	hiprandState_t localState = d_localstates[threadIdx.x];
	for (int i = 0; i < length; i++)
	{
		gamrnd_d(d_samples + sid + i, params, &localState);
	}
}

__global__ void betaTest_kernel(hiprandState_t* d_localstates, double2* params, double* d_samples, int length)
{
	int sid = threadIdx.x * length;
	hiprandState_t localState = d_localstates[threadIdx.x];
	for (int i = 0; i < length; i++)
	{
		betarnd_d(d_samples + sid + i, params, &localState);
	}
}

__global__ void initModelParams_kernel(_modelParams* modelParams, hiprandState_t* d_localstates)
{
	double x = hiprand_normal(&d_localstates[0]);
	double u = hiprand_uniform(&d_localstates[0]);
}

__global__ void sampleModelParams_kernel(_modelParams* modelParams, hiprandState_t* d_localstates)
{

}


void testRand()
{
	hiprandState_t* d_states;
	int seqs = 200;
	int length = 32;
	hipMalloc(&d_states, sizeof(hiprandState_t) * seqs);
	setup_kernel<<<1, seqs>>>(d_states);
	double* samples = new double[seqs * length];
	double* d_samples;
	size_t bytes = sizeof(double)*seqs * length;
	hipMalloc(&d_samples, bytes);
	hipMemset(d_samples, 0, bytes);
	/*generate_kernel<<<1, seqs>>>(d_samples, d_states);
	hipMemcpy(samples, d_samples, bytes, hipMemcpyDeviceToHost);

	for (int s = 0; s < seqs; s++)
	{
	for (int i = 0; i < length; i++)
	{
	printf("%4.2f ", samples[s * length + i]);
	}
	cout << endl;
	}*/


	{
		cout << "Gamma Distro" << endl;
		double2 params{ 0.5, 4 };
		double2* params_d;
		hipMalloc(&params_d, sizeof(double2));
		hipMemcpy(params_d, &params, sizeof(double2), hipMemcpyHostToDevice);
		gammaTest_kernel << <1, seqs >> >(d_states, params_d, d_samples, length);

		hipMemcpy(samples, d_samples, bytes, hipMemcpyDeviceToHost);

		/*for (int s = 0; s < seqs; s++)
		{
			for (int i = 0; i < length; i++)
			{
				printf("%4.2f ", samples[s * length + i]);
			}
			cout << endl;
		}*/

		ofstream gcsv("gamout.csv");
		for (int s = 0; s < seqs; s++)
		{
			for (int i = 0; i < length; i++)
			{
				gcsv << samples[s * length + i] << endl;
			}
		}
		gcsv.close();
	}

	{
		cout << "Beta Distro" << endl;
		double2 params{ 0.8, 4 };
		double2* params_d;
		hipMalloc(&params_d, sizeof(double2));
		hipMemcpy(params_d, &params, sizeof(double2), hipMemcpyHostToDevice);
		betaTest_kernel << <1, seqs >> >(d_states, params_d, d_samples, length);

		hipMemcpy(samples, d_samples, bytes, hipMemcpyDeviceToHost);

		ofstream gcsv("betaout.csv");
		for (int s = 0; s < seqs; s++)
		{
			for (int i = 0; i < length; i++)
			{
				gcsv << samples[s * length + i] << endl;
			}
		}
		gcsv.close();
	}

	// Free mems
	delete[] samples;
	hipFree(d_samples);
	hipFree(d_states);

}

void DLCode()
{
	int propImSize = 256;
	int propPatchSize = 8;
	int propImCount = 5;
	int N = calcN(propImSize, propPatchSize, propImCount);
	int M = propPatchSize*propPatchSize;
	int K = 100;
	cout << "M: " << M << ", N: " << N << ", K: " << K << endl;

	ImLoader imloader(propImSize, propPatchSize);
	gpuMat<double> Y(M, N);
	imloader.GetDataMatrix(Y, propImCount);

	DLConfig config1;
	DLConfig *config1_d;
	hipMalloc(&config1_d, sizeof(DLConfig));
	hipMemcpy(config1_d, &config1, sizeof(DLConfig), hipMemcpyHostToDevice);

	gpuMat<double> D(M, K);
	gpuMat<double> S(K, N);
	gpuMat<bool> B(K, N);
	gpuMat<double> PI(K, 1);
	gpuMat<double> post_PI(K, N);

	ModelParams modelParams1;

	
}