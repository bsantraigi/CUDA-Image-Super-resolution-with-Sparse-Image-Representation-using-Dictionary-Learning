#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#include <iostream>
#include "gpuMat.h"
#include "gpuOpsAPI.h"

#define pn(x) printf("%5.0f", (double)x)
#define min(x, y) (x)<(y)?(x):(y)

using namespace std;

int main()
{
	int S = 5;
	gpuMat<float> Y(S, S);
	gpuMat<int> B(S, S);
	gpuMat<double> C(S, S);
	cout << Y.cols << "by" << Y.rows << endl;

	for (int i = 0; i < S; i++)
	{
		for (int j = 0; j < S; j++)
		{
			Y(i, j) = i*Y.cols + j;
			B(i, j) = i>=j;
		}
	}
	Y.copy2Device();
	B.copy2Device();

	

	cout << endl;
	for (int i = 0; i < min(10, S); i++)
	{
		for (int j = 0; j < min(10, S); j++)
		{
			pn(Y(i, j));
		}
		cout << endl;
	}

	cout << endl;
	for (int i = 0; i < min(10, S); i++)
	{
		for (int j = 0; j < min(10, S); j++)
		{
			pn(B(i, j));

		}
		cout << endl;
	}

	// CUBLAS TEST
	/*float al = 1;
	float bet = 0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, S, &al, Y.d_elems, S, B.d_elems, S, &bet, C.d_elems, S);*/

	Add<float, int, double>(Y.d_elems, B.d_elems, C.d_elems, S, S);

	C.copy2Host();

	cout << endl;
	for (int i = 0; i < min(10, S); i++)
	{
		for (int j = 0; j < min(10, S); j++)
		{
			pn(C(i, j));
		}
		cout << endl;
	}
}