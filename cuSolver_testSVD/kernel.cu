#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
// Macro for timing kernel runs
#define START_METER {\
	hipEvent_t start, stop;\
	float elapsedTime;\
	hipEventCreate(&start);\
	hipEventRecord(start, 0);
#define STOP_METER hipEventCreate(&stop);\
	hipEventRecord(stop, 0);\
	hipEventSynchronize(stop);\
	hipEventElapsedTime(&elapsedTime, start, stop);\
	printf("Elapsed time : %f ms\n", elapsedTime);\
				}

int main()
{
	const int m = 64;
	const int rows = m;
	const int cols = m;
	/*       | 3.5 0.5 0 |
	*   A = | 0.5 3.5 0 |
	*       | 0   0   2 |
	*
	*/
	double A[rows*m];
	for (int i = 0; i < cols; i++)
	{
		for (int j = 0; j < rows; j++)
		{
			A[i*rows + j] = (double)rand() / RAND_MAX;
			if (i == j){
				A[i*rows + j] += 1;
			}
		}
	}

	hipsolverHandle_t handle;
	hipsolverDnCreate(&handle);
	int lwork;

	hipsolverDnDgesvd_bufferSize(
		handle,
		rows,
		cols,
		&lwork);

	double *d_A;
	hipMalloc(&d_A, sizeof(double)*rows*cols);
	hipMemcpy(d_A, A, sizeof(double)*rows*cols, hipMemcpyHostToDevice);

	double *d_S;
	hipMalloc(&d_S, sizeof(double)*rows);

	double *d_U;
	hipMalloc(&d_U, sizeof(double)*rows*rows);

	double *d_VT;
	hipMalloc(&d_VT, sizeof(double)*rows*rows);

	double *d_work;
	hipMalloc(&d_work, sizeof(double)*lwork);

	double *d_rwork;
	hipMalloc(&d_rwork, sizeof(double)*(rows - 1));

	int *devInfo;
	hipMalloc(&devInfo, sizeof(int));

	signed char jobu = 'A';
	signed char jobvt = 'A';
	START_METER
	hipsolverDnDgesvd(
		handle,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		rows,
		d_S,
		d_U,
		rows,
		d_VT,
		rows,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	STOP_METER
	hipFree(d_A);
	hipFree(d_rwork);
	hipFree(d_S);
	hipFree(d_U);
	hipFree(d_VT);
	hipFree(d_work);

}