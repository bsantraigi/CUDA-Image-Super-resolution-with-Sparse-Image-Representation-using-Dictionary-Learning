/*
* How to compile (assume cuda is installed at /usr/local/cuda/)
*   nvcc -c -I/usr/local/cuda/include syevd_example.cpp
*   g++ -o -fopenmp a.out syevd_example.o -L/usr/local/cuda/lib64 -lcudart -lcublas -lcusolver
*
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
// Macro for timing kernel runs
#define START_METER {\
	hipEvent_t start, stop;\
	float elapsedTime;\
	hipEventCreate(&start);\
	hipEventRecord(start, 0);
#define STOP_METER hipEventCreate(&stop);\
	hipEventRecord(stop, 0);\
	hipEventSynchronize(stop);\
	hipEventElapsedTime(&elapsedTime, start, stop);\
	printf("Elapsed time : %f ms\n", elapsedTime);\
		}
void printMatrix(int m, int n, const double*A, int rows, const char* name)
{
	for (int row = 0; row < m; row++){
		for (int col = 0; col < n; col++){
			double Areg = A[row + col*rows];
			printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
		}
	}
}

int main(int argc, char*argv[])
{
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	const int m = 64;
	const int rows = m;
	const int cols = m;
	/*       | 3.5 0.5 0 |
	*   A = | 0.5 3.5 0 |
	*       | 0   0   2 |
	*
	*/
	double A[rows*m];
	for (int i = 0; i < cols; i++)
	{
		for (int j = 0; j < rows; j++)
		{
			A[i*rows + j] = (double)rand() / RAND_MAX;
			if (i == j){
				A[i*rows + j] += 1;
			}
		}
	}
	//double lambda[m] = { 2.0, 3.0, 4.0 };

	double V[rows*m]; // eigenvectors
	double W[m]; // eigenvalues

	double *d_A = NULL;
	double *d_W = NULL;
	int *devInfo = NULL;
	double *d_work = NULL;
	int  lwork = 0;

	int info_gpu = 0;

	/*printf("A = (matlab base-1)\n");
	printMatrix(m, m, A, rows, "A");
	printf("=====\n");*/
		// step 1: create cusolver/cublas handle
		cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	cudaStat1 = hipMalloc((void**)&d_A, sizeof(double) * rows * m);
	cudaStat2 = hipMalloc((void**)&d_W, sizeof(double) * m);
	cudaStat3 = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * rows * m, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);

	// step 3: query working space of syevd
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	cusolver_status = hipsolverDnDsyevd_bufferSize(
		cusolverH,
		jobz,
		uplo,
		m,
		d_A,
		rows,
		d_W,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
	assert(hipSuccess == cudaStat1);

	// step 4: compute spectrum
	START_METER
	cusolver_status = hipsolverDnDsyevd(
		cusolverH,
		jobz,
		uplo,
		m,
		d_A,
		rows,
		d_W,
		d_work,
		lwork,
		devInfo);
	STOP_METER
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat1);

	cudaStat1 = hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(V, d_A, sizeof(double)*rows*m, hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	printf("after syevd: info_gpu = %d\n", info_gpu);
	assert(0 == info_gpu);

	/*printf("eigenvalue = (matlab base-1), ascending order\n");
	for (int i = 0; i < m; i++){
		printf("W[%d] = %E\n", i + 1, W[i]);
	}*/

	/*printf("V = (matlab base-1)\n");
	printMatrix(m, m, V, rows, "V");
	printf("=====\n");*/

	// step 4: check eigenvalues
	/*double lambda_sup = 0;
	for (int i = 0; i < m; i++){
		double error = fabs(lambda[i] - W[i]);
		lambda_sup = (lambda_sup > error) ? lambda_sup : error;
	}
	printf("|lambda - W| = %E\n", lambda_sup);*/

	// free resources
	if (d_A) hipFree(d_A);
	if (d_W) hipFree(d_W);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);

	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();

	return 0;
}